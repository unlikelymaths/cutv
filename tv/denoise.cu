#include "hip/hip_runtime.h"
#include "denoise.h"

#include "../util/eigs.h"
#include "../operators/derivative.h"
#include "kernels.h"

template<typename T>
TV::Denoise<T>::Denoise<T>(size_t width, size_t height)
	: TV::Base<T>(width, height)
{
	// Initialize
	hipMalloc(&df, width * height * sizeof(T));
	hipMalloc(&du, width * height * sizeof(T));
	hipMalloc(&du2, width * height * sizeof(T));
	hipMalloc(&dubar, width * height * sizeof(T));
	hipMalloc(&dp, 2 * width * height * sizeof(T));
	hipMalloc(&dTmp, width * height * sizeof(T));
	hipMalloc(&dTmp2, 2 * width * height * sizeof(T));

	// Compute stepsize
	op::DTD<T> op(width, height);
	T ev = eigenvalue<T>(&op);
	sigma = 0.8 / (ev);
	tau = sigma;
	alpha = 0.1f;

	// Kernel threads and blocks
	threads = dim3(32, 32, 1);
	blocks = dim3(iDivUp(width, 32), iDivUp(height, 32), 1);
	hipblasCreate(&handle);
}

template<typename T>
TV::Denoise<T>::~Denoise()
{
	hipFree(du);
	hipFree(du2);
	hipFree(dubar);
	hipFree(dp);
	hipFree(df);
	hipFree(dTmp);
	hipFree(dTmp2);
	hipblasDestroy(handle);
}

template<typename T>
void TV::Denoise<T>::apply(T* hImage)
{
	// Copy input image
	hipMemcpy(df, hImage, width * height * sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(du, df, width * height * sizeof(T), hipMemcpyDeviceToDevice);
	hipMemcpy(du2, df, width * height * sizeof(T), hipMemcpyDeviceToDevice);
	hipMemcpy(dubar, df, width * height * sizeof(T), hipMemcpyDeviceToDevice);

	// Initial P
	tvInitP KERNEL_ARGS2(blocks, threads) (dp, dubar, width, height);
	tvLimitP KERNEL_ARGS2(blocks, threads) (dp, alpha, width, height);

	// iterate
	int iterCount = 1000;
	for (int iter = 0; iter < iterCount; ++iter)
	{
		if (iter % 100 == 0) {
			computeEnergy();
		}
		// Update p
		tvUpdateP KERNEL_ARGS2(blocks, threads) (dp, dubar, sigma, width, height);
		tvLimitP KERNEL_ARGS2(blocks, threads) (dp, alpha, width, height);

		// Update u
		tvUpdateU KERNEL_ARGS2(blocks, threads)(du, du2, dp, df, tau, width, height);

		// Update u bar
		tvExterpolate KERNEL_ARGS2(blocks, threads)(du, du2, dubar, width, height);
	}
	hipMemcpy(hImage, dubar, width * height * sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
void TV::Denoise<T>::computeEnergy()
{
	T hEnergy[2] = { 0 };
	hipMemcpy(dTmp, du, width * height * sizeof(T), hipMemcpyDeviceToDevice);

	T a = -1;
	cublasaxpy(handle, width * height, &a, df, 1, dTmp, 1);
	cublasDot(handle, width * height, dTmp, 1, dTmp, 1, &hEnergy[0]);

	op::Derivative<T> derivative(width, height);
	derivative.apply(dTmp2, du);
	norm KERNEL_ARGS2(blocks, threads) (dTmp2, width, height);
	cublasasum(handle, width * height, dTmp2, 1, &hEnergy[1]);

	float energy = hEnergy[0] + alpha * hEnergy[1];
	std::cout << "Energy: " << energy << "\t\t(" << hEnergy[0] << " - " << alpha * hEnergy[1] << ")\n";
}


template class TV::Denoise<float>;
template class TV::Denoise<double>;