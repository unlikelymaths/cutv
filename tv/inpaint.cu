#include "hip/hip_runtime.h"
#include "inpaint.h"

#include "../util/eigs.h"
#include "../operators/derivative.h"
#include "kernels.h"

template<typename T>
TV::Inpaint<T>::Inpaint<T>(size_t width, size_t height, bool* mask)
	: TV::Base<T>(width, height)
{
	// Initialize
	hipMalloc(&df, width * height * sizeof(T));
	hipMalloc(&du, width * height * sizeof(T));
	hipMalloc(&du2, width * height * sizeof(T));
	hipMalloc(&dubar, width * height * sizeof(T));
	hipMalloc(&dp, 2 * width * height * sizeof(T));
	hipMalloc(&dTmp2, 2 * width * height * sizeof(T));
	hipMalloc(&dmask, width * height * sizeof(bool));
	hipMemcpy(dmask, mask, width * height * sizeof(bool), hipMemcpyHostToDevice);

	// Compute stepsize
	op::DTD<T> op(width, height);
	T ev = eigenvalue<T>(&op);
	sigma = 0.8 / (ev);
	tau = sigma;
	alpha = 1.f;

	// Kernel threads and blocks
	threads = dim3(32, 32, 1);
	blocks = dim3(iDivUp(width, 32), iDivUp(height, 32), 1);
	hipblasCreate(&handle);
}

template<typename T>
TV::Inpaint<T>::~Inpaint()
{
	hipFree(du);
	hipFree(du2);
	hipFree(dubar);
	hipFree(dp);
	hipFree(df);
	hipFree(dTmp2);
	hipFree(dmask);
	hipblasDestroy(handle);
}

template<typename T>
void TV::Inpaint<T>::apply(T* hImage)
{
	// Copy input image
	hipMemcpy(df, hImage, width * height * sizeof(T), hipMemcpyHostToDevice);
	hipMemcpy(du, df, width * height * sizeof(T), hipMemcpyDeviceToDevice);
	hipMemcpy(du2, df, width * height * sizeof(T), hipMemcpyDeviceToDevice);
	hipMemcpy(dubar, df, width * height * sizeof(T), hipMemcpyDeviceToDevice);

	// Initial P
	tvInitP KERNEL_ARGS2(blocks, threads) (dp, dubar, width, height);
	tvLimitP KERNEL_ARGS2(blocks, threads) (dp, alpha, width, height);

	// iterate
	int iterCount = 2000;
	for (int iter = 0; iter < iterCount; ++iter)
	{
		if (iter % 100 == 0) {
			computeEnergy();
		}
		// Update p
		tvUpdateP KERNEL_ARGS2(blocks, threads) (dp, dubar, sigma, width, height);
		tvLimitP KERNEL_ARGS2(blocks, threads) (dp, alpha, width, height);

		// Update u
		inpaintingUpdateU KERNEL_ARGS2(blocks, threads)(du, du2, dp, df, dmask, tau, width, height);

		// Update u bar
		tvExterpolate KERNEL_ARGS2(blocks, threads)(du, du2, dubar, width, height);
	}
	hipMemcpy(hImage, dubar, width * height * sizeof(T), hipMemcpyDeviceToHost);
}

template<typename T>
void TV::Inpaint<T>::computeEnergy()
{
	T hEnergy[1] = { 0 };

	op::Derivative<T> derivative(width, height);
	derivative.apply(dTmp2, du);
	norm KERNEL_ARGS2(blocks, threads) (dTmp2, width, height);
	cublasasum(handle, width * height, dTmp2, 1, &hEnergy[0]);

	std::cout << "Energy: " << hEnergy[0] << "\n";
}


template class TV::Inpaint<float>;
template class TV::Inpaint<double>;